#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include "mX_real.hpp"
using namespace mX_real;

template < typename T, T_mX(T) >
__device__ void bench( T * const  __restrict__ out_ ) {
  using TT = typename T::base_T;
  auto const ZERO = T(0);
  auto const ONE  = T(1);
  auto const TWO  = T(2);

  auto y = ZERO;
  auto s1 = 1;
  auto s2 = 2;
  auto s3 = 3;
  double f = 1.0;

  #pragma unroll(1)
  while ( true ) {
    if ( f == 0 ) break;

    auto z = ZERO;
    double h = 1.0;
    #pragma unroll(1)
    for(int i=0; i<9; i++) {

      z += ( ( TWO / s1 + TWO / s2 + ONE / s3 ) * h );

      s1 += 4;
      s2 += 4;
      s3 += 4;
      h *= double(-0.25);
    }
    y += z * f;

    f *= h;
  }

  *out_ = y;
}

template < typename T >
__global__ void benchmark( T * const  __restrict__ out_ ) {
  bench< T > ( out_ );
}

int main( int argc, char *argv[] ) {
  dd_Real_weakaccurate *x = nullptr;
  benchmark <<< 1, 1 >>> ( x );
}

