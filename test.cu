#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include "mX_real.hpp"
using namespace mX_real;

template < typename T, T_mX(T) >
__device__ void bench( int const n, T const * const x, T const * const y, T * const out_ ) {

  using TT = typename T::base_T;
  T const ZERO = T(0);
  T const ONE = T(1);
  T const TWO = T(2);
  T const THREE = T(3);
  T const FOUR = T(4);
  T const MQUATER = T(-0.25);

#if 0
  T z = ZERO;
  #pragma unroll(1)
  for(int i=0; i<n; i++) {
    z += x[i]*y[i];
  }
  *out_ = z;
#endif

  //
  {
    auto y = ZERO;
    auto t1 = ONE;
    auto t2 = TWO;
    auto t3 = THREE;
    auto t4 = ONE;
    #pragma unroll(1)
    for(int i=0; i<=400; i++) {
      auto z = ( TWO / t1 + TWO / t2 + ONE / t3 ) * t4;
      if ( z == ZERO ) break; // uderflow
      y += z;

      t1 += 4;
      t2 += 4;
      t3 += 4;
      t4 /= (-4);
    }
    *out_ = y;
  }

}

template < typename T >
__global__ void benchmark( int const n, T const * const x, T const * const y, T * const out_ ) {
  bench< T > ( n, x, y, out_ );
}

int main( int argc, char *argv[] ) {
  int n = 0;
  df_Real_sloppy *x = nullptr;
  df_Real_sloppy *y = nullptr;
  df_Real_sloppy *z = nullptr;
  std::cin >> n;
  benchmark <<< 1, 1 >>> ( n, x, y, z );
}

